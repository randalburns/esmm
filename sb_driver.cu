#include "sib_gpu.h"
#include "esmm_cpu.h"

#define cudaCheckErrors(msg) \
  do { \
    hipError_t __err = hipGetLastError(); \
    if (__err != hipSuccess) { \
        fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
            msg, hipGetErrorString(__err), \
            __FILE__, __LINE__); \
        fprintf(stderr, "*** FAILED - ABORTING\n"); \
        exit(1); \
    } \
  } while (0)


int main() {

    // Define 4x4 matrices A and B, and an output matrix C
    constexpr int rows = 4;
    constexpr int columns = 4;
    constexpr int inners = 4;

    size_t Asize = rows * inners * sizeof(float);
    size_t Bsize = inners * columns * sizeof(float);
    size_t Csize = rows * columns * sizeof(float);

     // Initialize matrices A and B with some values
    float A[rows * inners] = {1.0, 1.1, 1.2, 1.3,
                              2.0, 2.1, 2.2, 2.3,
                              3.0, 3.1, 3.2, 3.3,
                              4.0, 4.1, 4.2, 4.3};

    float B[inners * columns] = {1.0, 1.2, 1.4, 1.6,
                              2.0, 2.2, 2.4, 2.6,
                              3.0, 3.2, 3.4, 3.6,
                              4.0, 4.2, 4.4, 4.6};
	    
    float C[rows * columns];
    
    dim3 gridDim(1,1);
    dim3 blockDim(4,4);

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, Asize);
    hipMalloc((void **)&d_B, Bsize);
    hipMalloc((void **)&d_C, Csize);
    
    // Copy data from host to device
    hipMemcpy(d_A, A, Asize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, Bsize, hipMemcpyHostToDevice);

    // Zero target data
    hipMemset(d_C, 0, Csize);

    // Launch kernel
    esmm_naive<<<gridDim, blockDim>>>(rows, columns, inners, d_A, d_B, d_C);

    // Copy result from device to host
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    printf("\n Base \n\n");
    printMatrix<rows, columns>(C);

    // Sequential
    zeroMatrix<rows,columns>(C);
    hipMemset(d_C, 0, Csize);
    esmm_sequential<<<dim3(1,1), 4 * 4>>>(rows, columns, inners, 4, d_A, d_B, d_C);
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    printf("\n Sequential -- 4x4 \n\n");
    printMatrix<rows, columns>(C);

    // Sequential tiled
    zeroMatrix<rows,columns>(C);
    hipMemset(d_C, 0, Csize);
    esmm_sequential<<<dim3(2,2), 2*2>>>(rows, columns, inners, 2, d_A, d_B, d_C);
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    printf("\n Sequential -- 2x2 \n\n");
    printMatrix<rows, columns>(C);

    // shared memory
    zeroMatrix<rows,columns>(C);
    hipMemset(d_C, 0, Csize);
    esmm_shmem<<<dim3(1,1), dim3(4*4), 4*4*2*sizeof(float)>>>(rows, columns, inners, 4, d_A, d_B, d_C);
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    printf("\n Sequential shared memory \n\n");
    printMatrix<rows, columns>(C);

    // shared memory tiled
    zeroMatrix<rows,columns>(C);
    hipMemset(d_C, 0, Csize);
    esmm_shmem<<<dim3(2,2), dim3(2*2), 2*2*2*sizeof(float)>>>(rows, columns, inners, 2, d_A, d_B, d_C);
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    printf("\n Sequential shared memory -- 2x2 \n\n");
    printMatrix<rows, columns>(C);

    // sb shared memory
    zeroMatrix<rows,columns>(C);
    hipMemset(d_C, 0, Csize);
    sb_shmem<<<dim3(1,1), dim3(4*4), 4*4*2*sizeof(float)>>>(rows, columns, inners, 4, d_A, d_B, d_C);
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    printf("\n Siboehm shared memory \n\n");
    printMatrix<rows, columns>(C);

    // sb shared memory tiled
    zeroMatrix<rows,columns>(C);
    hipMemset(d_C, 0, Csize);
    sb_shmem<<<dim3(2,2), dim3(2*2), 2*2*2*sizeof(float)>>>(rows, columns, inners, 2, d_A, d_B, d_C);
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    printf("\n Siboehm shared memory -- 2x2 \n\n");
    printMatrix<rows, columns>(C);

    // sb 1d warp 
    //
    //   16 cells / TM threads 
    zeroMatrix<rows,columns>(C);
    hipMemset(d_C, 0, Csize);
    sb_1dwarp_tile<<<dim3(1,1), dim3(4*4/TM), 4*4*2*sizeof(float)>>>(rows, columns, inners, 
		    							4, 4, 4/TM, d_A, d_B, d_C);
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    printf("\n Siboehm 1-d warp \n\n");
    printMatrix<rows, columns>(C);

    // sb 1d warp tiled
    //
    //   4 cells / TM threads 
    zeroMatrix<rows,columns>(C);
    hipMemset(d_C, 0, Csize);
    sb_1dwarp_tile<<<dim3(2,2), dim3(2*2/TM), 2*2*2*sizeof(float)>>>(rows, columns, inners, 
		    							2, 2, 2/TM, d_A, d_B, d_C);
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    printf("\n Siboehm 1-d warp tiled \n\n");
    printMatrix<rows, columns>(C);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
