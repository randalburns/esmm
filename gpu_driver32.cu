#include "hip/hip_runtime.h"
#include "esmm_gpu.h"
#include "esmm_cpu.h"


// Function to check if two matrices are equal within a tolerance
bool checkEqual(int rows, int cols, float* matrix1, float* matrix2, float tolerance = 1e-6) {
    for (int row = 0; row < rows; ++row) {
        for (int col = 0; col < cols; ++col) {
            if (std::fabs(matrix1[row * cols + col] - matrix2[row * cols + col]) > tolerance) {
                std::cout << std::endl;
                std::cout << "Value1 " << matrix1[row * cols + col] 
                          << " Value2 " << matrix2[row * cols + col] 
               	          << " offset " << row << ", " << col 
                          << " Value2 " << matrix2[row * cols + col] 
			  << " Difference  "  
                          << std::fabs(matrix1[row * cols + col] - matrix2[row * cols + col]) << std::endl;
                return false;
            }
        }
    }
    return true;
}


int main() {

    // Define 4x4 matrices A and B, and an output matrix C
    constexpr int rows = 32;
    constexpr int columns = 32;
    constexpr int inners = 32; 
     
    size_t Asize = rows * inners * sizeof(float);
    size_t Bsize = inners * columns * sizeof(float);
    size_t Csize = rows * columns * sizeof(float);

    float A[rows * inners];
    float B[inners * columns];
    float C[rows * columns];
    float Cref[rows * columns];
    
    // Initialize A to random floating-point values between 0 and 1
    for (int row = 0; row < rows; ++row) {
        for (int inner = 0; inner < inners; ++inner) {
            A[row * inners + inner] = static_cast<float>(rand()) / RAND_MAX;
        }
    }

    // Initialize B to random floating-point values between 0 and 1
    for (int inner = 0; inner < inners; ++inner) {
        for (int col = 0; col < columns; ++col) {
            B[inner * columns + col] = static_cast<float>(rand()) / RAND_MAX;
        }
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, Asize);
    hipMalloc((void **)&d_B, Bsize);
    hipMalloc((void **)&d_C, Csize);
    
    // Copy data from host to device
    hipMemcpy(d_A, A, Asize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, Bsize, hipMemcpyHostToDevice);

    // Tiled naive
    hipMemset(d_C, 0, Csize);
    zeroMatrix<rows,columns>(Cref);
    esmm_naive<<<dim3(1,1), dim3(32,32)>>>(rows, columns, inners, d_A, d_B, d_C);
    hipMemcpy(Cref, d_C, Csize, hipMemcpyDeviceToHost);
    hipMemset(d_C, 0, Csize);

    // Tiled naive
    hipMemset(d_C, 0, Csize);
    zeroMatrix<rows,columns>(C);
    esmm_naive<<<dim3(2,2), dim3(16,16)>>>(rows, columns, inners, d_A, d_B, d_C);
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    std::cout << "Naive tiled kernel matches = " << checkEqual ( rows, columns, Cref, C ) << std::endl;
    hipMemset(d_C, 0, Csize);
     
    // sequential
    hipMemset(d_C, 0, Csize);
    zeroMatrix<rows,columns>(C);
    esmm_sequential<<<dim3(1,1), 32*32>>>(rows, columns, inners, 32, d_A, d_B, d_C);
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    std::cout << "Sequential kernel matches = " << checkEqual ( rows, columns, Cref, C ) << std::endl;
    hipMemset(d_C, 0, Csize);

    // tiled sequential
    hipMemset(d_C, 0, Csize);
    zeroMatrix<rows,columns>(C);
    esmm_sequential<<<dim3(2,2), 16*16>>>(rows, columns, inners, 16, d_A, d_B, d_C);
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    std::cout << "Sequential tiled kernel matches = " << checkEqual ( rows, columns, Cref, C ) << std::endl;
    hipMemset(d_C, 0, Csize);

    // shared memory
    hipMemset(d_C, 0, Csize);
    zeroMatrix<rows,columns>(C);
    esmm_sequential_shmem<<<dim3(1,1), dim3(32*32), 32*32*2>>>(rows, columns, inners, 32, d_A, d_B, d_C);
//    esmm_sequential_shmem<<<gridDim, blockDim.x * blockDim.y, 2 * blockDim.x * blockDim.y>>>(rows, columns, inners, blockDim.x, d_A, d_B, d_C);
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    std::cout << "Shared memory kernel = " << checkEqual ( rows, columns, Cref, C ) << std::endl;
    hipMemset(d_C, 0, Csize);

    // tiled shared memory
    hipMemset(d_C, 0, Csize);
    zeroMatrix<rows,columns>(C);
    esmm_sequential_shmem<<<dim3(2,2), 16*16, 16*16*2>>>(rows, columns, inners, 16, d_A, d_B, d_C);
//    esmm_sequential_shmem<<<gridDim, blockDim.x * blockDim.y, 2 * blockDim.x * blockDim.y>>>(rows, columns, inners, blockDim.x, d_A, d_B, d_C);
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    std::cout << "Tiled shared memory kernel = " << checkEqual ( rows, columns, Cref, C ) << std::endl;
    hipMemset(d_C, 0, Csize);
     
    return;

    // multi
    hipMemset(d_C, 0, Csize);
    zeroMatrix<rows,columns>(C);
    esmm_shmem_multi<<<gridDim, blockDim.x, 2 * blockDim.x * blockDim.y>>>(rows, columns, inners, blockDim.x, d_A, d_B, d_C);
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    std::cout << "Tiled multi matches = " << checkEqual ( rows, columns, Cref, C ) << std::endl;
    hipMemset(d_C, 0, Csize);

    // multi2
    hipMemset(d_C, 0, Csize);
    zeroMatrix<rows,columns>(C);
    esmm_shmem_multi<<<gridDim, blockDim.x, 2 * blockDim.x * blockDim.y>>>(rows, columns, inners, blockDim.x, d_A, d_B, d_C);
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    std::cout << "Tiled multi 2 matches = " << checkEqual ( rows, columns, Cref, C ) << std::endl;
    hipMemset(d_C, 0, Csize);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
