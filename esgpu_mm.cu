#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>
#include "esmm_cpu.h"

__global__ void esmm_naive(int rows, int cols, int inners, const float *A,
                           const float *B, float *C)
{
    // compute position in C that this thread is responsible for
    int xidx = blockIdx.x * blockDim.x + threadIdx.x;
    int yidx = blockIdx.y * blockDim.y + threadIdx.y;

    // along rows and colums of AB for each element in C
    int i=0;
    float tmp = 0.0;
    for (; i < inners; ++i)
    {
	  // sums that work as a test/placeholder
          // Row of A
          //tmp += A[xidx * cols + i]; 
          // Col of B
          // tmp += B[i * inners + yidx]; 
	  // Row of A  Col of B Sum
          // tmp += A[xidx * cols + i] + B[i * inners + yidx]; 

	  // Multiply works on full grid
          tmp += A[xidx * cols + i] * B[i * inners + yidx]; 
    }
    C[xidx * cols + yidx] = tmp;
}

int main() {

    // Define 4x4 matrices A and B, and an output matrix C
    constexpr int rows = 4;
    constexpr int columns = 4;
    constexpr int inners = 4;

    size_t Asize = rows * inners * sizeof(float);
    size_t Bsize = inners * columns * sizeof(float);
    size_t Csize = rows * columns * sizeof(float);

     // Initialize matrices A and B with some values
    float A[rows * inners] = {1.0, 1.1, 1.2, 1.3,
                              2.0, 2.1, 2.2, 2.3,
                              3.0, 3.1, 3.2, 3.3,
                              4.0, 4.1, 4.2, 4.3};

    float B[inners * columns] = {1.0, 1.2, 1.4, 1.6,
                              2.0, 2.2, 2.4, 2.6,
                              3.0, 3.2, 3.4, 3.6,
                              4.0, 4.2, 4.4, 4.6};
	    
    float C[rows * columns];
    zeroMatrix<rows,columns>(C);
    
    dim3 gridDim(1,1);
    dim3 blockDim(4,4);

    // create as many blocks as necessary to map all of C
    // dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32), 1);
    // 32 * 32 = 1024 thread per block
    // dim3 blockDim(32, 32, 1);
	
    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, Asize);
    hipMalloc((void **)&d_B, Bsize);
    hipMalloc((void **)&d_C, Csize);
    
    // Copy data from host to device
    hipMemcpy(d_A, A, Asize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, Bsize, hipMemcpyHostToDevice);

    printf("\n A \n\n");
    printMatrix<rows, columns>(A);
    printf("\n B \n\n");
    printMatrix<rows, columns>(B);

    printf("\n Output \n\n");

    // Launch kernel
    esmm_naive<<<gridDim, blockDim>>>(rows, columns, inners, d_A, d_B, d_C);

    // Copy result from device to host
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);

    printMatrix<rows, columns>(C);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
