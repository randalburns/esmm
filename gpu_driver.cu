#include "esmm_gpu.h"
#include "esmm_cpu.h"

int main() {

    // Define 4x4 matrices A and B, and an output matrix C
    constexpr int rows = 4;
    constexpr int columns = 4;
    constexpr int inners = 4;

    size_t Asize = rows * inners * sizeof(float);
    size_t Bsize = inners * columns * sizeof(float);
    size_t Csize = rows * columns * sizeof(float);

     // Initialize matrices A and B with some values
    float A[rows * inners] = {1.0, 1.1, 1.2, 1.3,
                              2.0, 2.1, 2.2, 2.3,
                              3.0, 3.1, 3.2, 3.3,
                              4.0, 4.1, 4.2, 4.3};

    float B[inners * columns] = {1.0, 1.2, 1.4, 1.6,
                              2.0, 2.2, 2.4, 2.6,
                              3.0, 3.2, 3.4, 3.6,
                              4.0, 4.2, 4.4, 4.6};
	    
    float C[rows * columns];
    
    dim3 gridDim(1,1);
    dim3 blockDim(4,4);

    // create as many blocks as necessary to map all of C
    // dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32), 1);
    // 32 * 32 = 1024 thread per block
    // dim3 blockDim(32, 32, 1);
	
    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, Asize);
    hipMalloc((void **)&d_B, Bsize);
    hipMalloc((void **)&d_C, Csize);
    
    // Copy data from host to device
    hipMemcpy(d_A, A, Asize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, Bsize, hipMemcpyHostToDevice);

    // Zero target data
    hipMemset(d_C, 0, Csize);

//    printf("\n A \n\n");
//    printMatrix<rows, columns>(A);
//    printf("\n B \n\n");
//    printMatrix<rows, columns>(B);


    // Launch kernel
    esmm_naive<<<gridDim, blockDim>>>(rows, columns, inners, d_A, d_B, d_C);

    // Copy result from device to host
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    printf("\n Base \n\n");
    printMatrix<rows, columns>(C);

    // Zero target matrix
    hipMemset(d_C, 0, Csize);

    // Launch kernel
    esmm_sequential<<<gridDim, blockDim.x * blockDim.y>>>(rows, columns, inners, blockDim.x, d_A, d_B, d_C);
    //esmm_sequential<<<dim3(2,2), 4>>>(rows, inners, columns, 2, d_A, d_B, d_C);

    // Copy result from device to host
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    printf("\n Sequential \n\n");
    printMatrix<rows, columns>(C);


    // Zero target matrix
    hipMemset(d_C, 0, Csize);

    // Launch kernel
    //esmm_Btile<<<dim3(1,1), dim3(4)>>>(rows, columns, inners, 4, 4, d_A, d_B, d_C);
    //esmm_Btile<<<dim3(2,2), dim3(2)>>>(rows, columns, inners, 2, 2, d_A, d_B, d_C);
    
   esmm_Btile_noatomic<<<dim3(1,1), dim3(4)>>>(rows, columns, inners, 4, 4, d_A, d_B, d_C);
//    esmm_Btile_noatomic<<<dim3(2,2), dim3(2)>>>(rows, columns, inners, 2, 2, d_A, d_B, d_C);
    // Copy result from device to host
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);

    printf("\n Tiled \n\n");
    printMatrix<rows, columns>(C);

    // Zero target matrix
    hipMemset(d_C, 0, Csize);

    // Launch kernel
    esmm_Btile_shmem<<<dim3(1,1), dim3(4), 3*4*4*sizeof(float)>>>(rows, columns, inners, 4, 4, d_A, d_B, d_C);
    
    // Copy result from device to host
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);

    printf("\n Shmem \n\n");
    printMatrix<rows, columns>(C);

/* 
    // Zero target matrix
    hipMemset(d_C, 0, Csize);

    //esmm_tile_noatomic<<<dim3(1,1,1), dim3(1,1,4)>>>(rows, inners, columns, 4, 4, d_A, d_B, d_C);
    esmm_tile_noatomic<<<dim3(2,2,1), dim3(1,1,4)>>>(rows, inners, columns, 2, 2, d_A, d_B, d_C);

    // Copy result from device to host
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);

    printf("\n Tiled no atomic \n\n");
    printMatrix<rows, columns>(C);

*/
    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
