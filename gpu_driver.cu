#include "esmm_gpu.h"
#include "esmm_cpu.h"

int main() {

    // Define 4x4 matrices A and B, and an output matrix C
    constexpr int rows = 4;
    constexpr int columns = 4;
    constexpr int inners = 4;

    size_t Asize = rows * inners * sizeof(float);
    size_t Bsize = inners * columns * sizeof(float);
    size_t Csize = rows * columns * sizeof(float);

     // Initialize matrices A and B with some values
    float A[rows * inners] = {1.0, 1.1, 1.2, 1.3,
                              2.0, 2.1, 2.2, 2.3,
                              3.0, 3.1, 3.2, 3.3,
                              4.0, 4.1, 4.2, 4.3};

    float B[inners * columns] = {1.0, 1.2, 1.4, 1.6,
                              2.0, 2.2, 2.4, 2.6,
                              3.0, 3.2, 3.4, 3.6,
                              4.0, 4.2, 4.4, 4.6};
	    
    float C[rows * columns];
    
    dim3 gridDim(1,1);
    dim3 blockDim(4,4);

    // create as many blocks as necessary to map all of C
    // dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32), 1);
    // 32 * 32 = 1024 thread per block
    // dim3 blockDim(32, 32, 1);
	
    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, Asize);
    hipMalloc((void **)&d_B, Bsize);
    hipMalloc((void **)&d_C, Csize);
    
    // Copy data from host to device
    hipMemcpy(d_A, A, Asize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, Bsize, hipMemcpyHostToDevice);

    // Zero target data
    hipMemset(d_C, 0, Csize);

//    printf("\n A \n\n");
//    printMatrix<rows, columns>(A);
//    printf("\n B \n\n");
//    printMatrix<rows, columns>(B);


    // Launch kernel
    esmm_naive<<<gridDim, blockDim>>>(rows, inners, columns, d_A, d_B, d_C);

    // Copy result from device to host
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    printf("\n Base \n\n");
    printMatrix<rows, columns>(C);

    // Zero target matrix
    hipMemset(d_C, 0, Csize);

    // Launch kernel
    //esmm_nogrid<<<1, 4>>>(rows, inners, columns, d_A, d_B, d_C);
    //esmm_tile<<<dim3(1,1,1), dim3(1,1,4)>>>(rows, inners, columns, 4, 4, d_A, d_B, d_C);
    esmm_tile<<<dim3(2,2,1), dim3(1,1,4)>>>(rows, inners, columns, 2, 2, d_A, d_B, d_C);
    //esmm_tile<<<dim3(4,4,1), dim3(1,1,4)>>>(rows, inners, columns, 1, 1, d_A, d_B, d_C);
    
    // Copy result from device to host
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);

    printf("\n Threaded \n\n");
    printMatrix<rows, columns>(C);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
