#include "hip/hip_runtime.h"
#include "esmm_gpu.h"
#include "esmm_cpu.h"


// Function to check if two matrices are equal within a tolerance
bool checkEqual(int rows, int cols, float* matrix1, float* matrix2, float tolerance = 1e-6) {
    for (int row = 0; row < rows; ++row) {
        for (int col = 0; col < cols; ++col) {
            if (std::fabs(matrix1[row * cols + col] - matrix2[row * cols + col]) > tolerance) {
                std::cout << std::endl;
                std::cout << "Value1 " << matrix1[row * cols + col] 
                          << " Value2 " << matrix2[row * cols + col] 
               	          << " offset " << row << ", " << col 
                          << " Value2 " << matrix2[row * cols + col] 
			  << " Difference  "  
                          << std::fabs(matrix1[row * cols + col] - matrix2[row * cols + col]) << std::endl;
                return false;
            }
        }
    }
    return true;
}


// Only works for up to 32 in oneMatrix

int main() {

    // Define 4x4 matrices A and B, and an output matrix C
    constexpr int rows = 32;
    constexpr int columns = 32;
    constexpr int inners = 32; 
     
    // base grid
    dim3 oneGrid(1,1);
    dim3 fullBlock(rows,columns);

    dim3 gridDim(2,2);
    dim3 blockDim(16,16);

    // rectangular 2,4
    dim3 gridDim24(2,4);
    dim3 blockDim24(16,8);
    
    // rectangular 4,2
    dim3 gridDim42(4,2);
    dim3 blockDim42(8,16);

    size_t Asize = rows * inners * sizeof(float);
    size_t Bsize = inners * columns * sizeof(float);
    size_t Csize = rows * columns * sizeof(float);

    float A[rows * inners];
    float B[inners * columns];
    float C[rows * columns];
    float Cref[rows * columns];
    float Ccpu[rows * columns];
    
    // Initialize A to random floating-point values between 0 and 1
    for (int row = 0; row < rows; ++row) {
        for (int inner = 0; inner < inners; ++inner) {
            A[row * inners + inner] = static_cast<float>(rand()) / RAND_MAX;
        }
    }

    // Initialize B to random floating-point values between 0 and 1
    for (int inner = 0; inner < inners; ++inner) {
        for (int col = 0; col < columns; ++col) {
            B[inner * columns + col] = static_cast<float>(rand()) / RAND_MAX;
        }
    }

    // baseMM on CPU
    zeroMatrix<rows,columns>(Ccpu);
    baseMM<rows, columns, inners>(A, B, Ccpu);
	    
    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, Asize);
    hipMalloc((void **)&d_B, Bsize);
    hipMalloc((void **)&d_C, Csize);
    
    // Copy data from host to device
    hipMemcpy(d_A, A, Asize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, Bsize, hipMemcpyHostToDevice);

    // Whole matrix in one kernel -- this is reference Cref
    hipMemset(d_C, 0, Csize);
    esmm_naive<<<oneGrid, fullBlock>>>(rows, columns, inners, d_A, d_B, d_C);
    hipMemcpy(Cref, d_C, Csize, hipMemcpyDeviceToHost);
    hipMemset(d_C, 0, Csize);

    // Check against CPU -- need a higher tolerance than 1e-6
    std::cout << "Base matches CPU = " << checkEqual ( rows, columns, Ccpu, Cref, 0.001 ) << std::endl;

    // Tiled naive
    hipMemset(d_C, 0, Csize);
    esmm_naive<<<gridDim, blockDim>>>(rows, columns, inners, d_A, d_B, d_C);
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    std::cout << "Tiled naive kernel matches = " << checkEqual ( rows, columns, Cref, C ) << std::endl;
    hipMemset(d_C, 0, Csize);

    // tiled sequential
    hipMemset(d_C, 0, Csize);
    esmm_sequential<<<gridDim, blockDim.x * blockDim.y>>>(rows, columns, inners, blockDim.x, d_A, d_B, d_C);
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    std::cout << "Sequential kernel matches = " << checkEqual ( rows, columns, Cref, C ) << std::endl;
    hipMemset(d_C, 0, Csize);

    // tiled not square	
    hipMemset(d_C, 0, Csize);
    esmm_sequential_ns<<<gridDim24, blockDim24.x * blockDim24.y>>>(rows, columns, inners, blockDim24.x, blockDim24.y, d_A, d_B, d_C);
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    std::cout << "Not square 24 kernel matches = " << checkEqual ( rows, columns, Cref, C ) << std::endl;
    hipMemset(d_C, 0, Csize);

    // tiled not square	
    hipMemset(d_C, 0, Csize);
    esmm_sequential_ns<<<gridDim42, blockDim42.x * blockDim42.y>>>(rows, columns, inners, blockDim42.x, blockDim42.y, d_A, d_B, d_C);
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    std::cout << "Not square 42 kernel matches = " << checkEqual ( rows, columns, Cref, C ) << std::endl;
    hipMemset(d_C, 0, Csize);

    // tiled shared memory
    hipMemset(d_C, 0, Csize);
    esmm_sequential_shmem<<<gridDim, blockDim.x * blockDim.y, 2 * blockDim.x * blockDim.y>>>(rows, columns, inners, blockDim.x, d_A, d_B, d_C);
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    std::cout << "Tiled shared memory kernel = " << checkEqual ( rows, columns, Cref, C ) << std::endl;
    hipMemset(d_C, 0, Csize);


    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
