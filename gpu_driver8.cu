#include "esmm_gpu.h"
#include "esmm_cpu.h"

int main() {

    // Define 4x4 matrices A and B, and an output matrix C
    constexpr int rows = 8;
    constexpr int columns = 8;
    constexpr int inners = 8;

    size_t Asize = rows * inners * sizeof(float);
    size_t Bsize = inners * columns * sizeof(float);
    size_t Csize = rows * columns * sizeof(float);

     // Initialize matrices A and B with some values
    float A[rows * inners] = {1.0, 1.1, 1.2, 1.3, 1.4, 1.5, 1.6, 1.7,
                              2.0, 2.1, 2.2, 2.3, 2.4, 2.5, 2.6, 2.7,
                              3.0, 3.1, 3.2, 3.3, 3.4, 3.5, 3.6, 3.7,
                              4.0, 4.1, 4.2, 4.3, 4.4, 4.5, 4.6, 4.7,
    			      1.0, 1.1, 1.2, 1.3, 1.4, 1.5, 1.6, 1.7,
                              2.0, 2.1, 2.2, 2.3, 2.4, 2.5, 2.6, 2.7,
                              3.0, 3.1, 3.2, 3.3, 3.4, 3.5, 3.6, 3.7,
                              4.0, 4.1, 4.2, 4.3, 4.4, 4.5, 4.6, 4.7};

    float B[inners * columns] = {1.0, 1.1, 1.2, 1.3, 1.4, 1.5, 1.6, 1.7,
                              2.0, 2.1, 2.2, 2.3, 2.4, 2.5, 2.6, 2.7,
                              3.0, 3.1, 3.2, 3.3, 3.4, 3.5, 3.6, 3.7,
                              4.0, 4.1, 4.2, 4.3, 4.4, 4.5, 4.6, 4.7,
    			      1.0, 1.1, 1.2, 1.3, 1.4, 1.5, 1.6, 1.7,
                              2.0, 2.1, 2.2, 2.3, 2.4, 2.5, 2.6, 2.7,
                              3.0, 3.1, 3.2, 3.3, 3.4, 3.5, 3.6, 3.7,
                              4.0, 4.1, 4.2, 4.3, 4.4, 4.5, 4.6, 4.7};

    float C[rows * columns];
    
    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, Asize);
    hipMalloc((void **)&d_B, Bsize);
    hipMalloc((void **)&d_C, Csize);
    
    // Copy data from host to device
    hipMemcpy(d_A, A, Asize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, Bsize, hipMemcpyHostToDevice);

    // Zero target data
    hipMemset(d_C, 0, Csize);

    // Launch kernel
    esmm_naive<<<dim3(1,1), dim3(8,8)>>>(rows, columns, inners, d_A, d_B, d_C);

    // Copy result from device to host
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    printf("\n Base \n\n");
    printMatrix<rows, columns>(C);

    // Sequential
    zeroMatrix<rows,columns>(C);
    hipMemset(d_C, 0, Csize);
    esmm_sequential<<<dim3(1,1), dim3(8*8)>>>(rows, columns, inners, 8, d_A, d_B, d_C);
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    printf("\n Sequential -- 8x8 \n\n");
    printMatrix<rows, columns>(C);

    // Sequential tiled
    zeroMatrix<rows,columns>(C);
    hipMemset(d_C, 0, Csize);
    esmm_sequential<<<dim3(2,2), dim3(4*4)>>>(rows, columns, inners, 4, d_A, d_B, d_C);
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    printf("\n Sequential -- 4x4 \n\n");
    printMatrix<rows, columns>(C);

    // Sequential tiled
    zeroMatrix<rows,columns>(C);
    hipMemset(d_C, 0, Csize);
    esmm_sequential<<<dim3(4,4), 2*2>>>(rows, columns, inners, 2, d_A, d_B, d_C);
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    printf("\n Sequential -- 2x2 \n\n");
    printMatrix<rows, columns>(C);

    // shared memory
    zeroMatrix<rows,columns>(C);
    hipMemset(d_C, 0, Csize);
    esmm_sequential_shmem<<<dim3(1,1), dim3(8*8), 8*8*2>>>(rows, columns, inners, 8, d_A, d_B, d_C);
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    printf("\n Sequential shared memory -- 4x4 \n\n");
    printMatrix<rows, columns>(C);

    return;

    // shared memory tiled
    zeroMatrix<rows,columns>(C);
    hipMemset(d_C, 0, Csize);
    esmm_sequential_shmem<<<dim3(2,2), 2*2, 2*2*2>>>(rows, columns, inners, 2, d_A, d_B, d_C);
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    printf("\n Sequential shared memory -- 2x2 \n\n");
    printMatrix<rows, columns>(C);

    // multi 
    zeroMatrix<rows,columns>(C);
    hipMemset(d_C, 0, Csize);
    esmm_shmem_multi<<<dim3(1,1), 4, 4*4*2>>>(rows, columns, inners, 4, d_A, d_B, d_C);
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    printf("\n Multi -- 4x4 \n\n");
    printMatrix<rows, columns>(C);

    // multi tiled
    zeroMatrix<rows,columns>(C);
    hipMemset(d_C, 0, Csize);
    esmm_shmem_multi<<<dim3(2,2), 2, 2*2*2>>>(rows, columns, inners, 2, d_A, d_B, d_C);
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    printf("\n Multi tiled -- 2x2 \n\n");
    printMatrix<rows, columns>(C);

    // multi 
    zeroMatrix<rows,columns>(C);
    hipMemset(d_C, 0, Csize);
    esmm_shmem_multi2<<<dim3(1,1), 4, 4*4*2>>>(rows, columns, inners, 4, d_A, d_B, d_C);
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    printf("\n Multi 2 -- 4x4 \n\n");
    printMatrix<rows, columns>(C);

    // multi2 tiled
    zeroMatrix<rows,columns>(C);
    hipMemset(d_C, 0, Csize);
    esmm_shmem_multi2<<<dim3(2,2), 2, 2*2*2>>>(rows, columns, inners, 2, d_A, d_B, d_C);
    hipMemcpy(C, d_C, Csize, hipMemcpyDeviceToHost);
    printf("\n Multi 2 tiled -- 2x2 \n\n");
    printMatrix<rows, columns>(C);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;

}
